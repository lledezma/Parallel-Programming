    //adding two values using CUDA
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
__global__ void add(int *d_a, int *d_b, int *d_result) {
      *d_result = *d_a + *d_b;
}
int main() {
  //declarre host variables
  int h_a = 50;
  int h_b = 50;
  int h_result = 0;
  
  //declare device variables
  int *d_a;
  int *d_b;
  int *d_result;

  //Memory allocation of device variable
  hipMalloc((void**)&d_a, sizeof(int));
  hipMalloc((void**)&d_b, sizeof(int));
  hipMalloc((void**)&d_result, sizeof(int));

  //Copy Host memory to Device memory
  hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

  //Launch Kernel
  add<<<1,1>>>(d_a,d_b,d_result);

  //copy device results to host results
  hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

  printf("the result is: %d\n",h_result);

  //free device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_result);
  return 0;
}