#include "hip/hip_runtime.h"
			//Pattern Match Program using CUDA 
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void match(char* pattern, char* string, int* results, int patLength, int strLength){
	//get global thread id
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  //check bounds
  if (x <= strLength-patLength){                                                         
    for(int i = 0; i < patLength; i++){                                               
      if (pattern[i] != string[idx+i])   //if a mismatch is found, exit.        
        return;                                    
    }
    atomicAdd(results,1); //match has been found so we add 1 to results
  }       
}

int main(){
  //device variables
  char* d_pattern;       //device pattern
  char* d_string;        //device string
  int* d_results;        //device variable to store results

  //host variables
  int h_results;        //host varible to store results  
  const char* h_pattern = "NNTHVLTLP";
  const char* h_string = "MIVNNTHVLTLPLYTTTTCHTHPHLYTNNTHVLTLPYSIYHLKLTLLSDSTSLHGPSCHTHNNTHVLTLPTHVLTLLTLLSDSTSRWGSK";
  int h_patLength = (int)strlen(h_pattern); //length of pattern
  int h_strLength = (int)strlen(h_string);  //length of string

  //Memory allocation of device variables
  hipMalloc((void**)&d_pattern, h_patLength*sizeof(char));
  hipMalloc((void**)&d_string, h_strLength*sizeof(char));
  hipMalloc((void**)&d_results, sizeof(int));

  //Copy Host memory to Device Memory
  hipMemcpy(d_pattern,h_pattern, h_patLength*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(d_string,h_string, h_strLength*sizeof(char), hipMemcpyHostToDevice);

  //Launch Kernel
  match<<<1,h_strLength>>>(d_pattern,d_string,d_results,h_patLength, h_strLength);      //A grid with one block and strLength threads.

  //copy device results to host results
  hipMemcpy(&h_results, d_results, sizeof(int), hipMemcpyDeviceToHost);

  //print results 
  printf("Total number of matches: %d\n", h_results);

  //free device memory
  hipFree(d_pattern);
  hipFree(d_string);
  hipFree(d_results);

  return 0;
}
