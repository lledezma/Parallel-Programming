		//multiplication of two matrices using a kernel with a 2d grid and 2d blocks
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void multiMatrix(int* A, int* B, int*C, int colA, int colB, int rowA){
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

  	int sum=0;
	//check bounds
	if(x < colB && y < rowA){
		for(int i = 0; i < colA; i++){
			sum += A[y * colA + i] * B[i * colB + x];
		}
		C[y * colB + x] = sum;
	}
}

int main(){
	int BLOCK_SIZE = 16;
	//rows and columns 
	int rowA = 15;
	int colA = 15;
	int rowB = colA;
	int colB = 10;

	//Declaring host variables
	int h_A[colA*rowA], h_B[colB*rowB], h_C[colB*rowA];
	//Declaring device variables
	int *d_A,*d_B,*d_C;

	//Memory allocation of device variables
	hipMalloc((void**)&d_A, (colA*rowA)*sizeof(int));
	hipMalloc((void**)&d_B, (colB*rowB)*sizeof(int));
	hipMalloc((void**)&d_C, (colB*rowA)*sizeof(int));

	//initializing host matrices
	for(int i = 0; i < (colA*rowA); i++){
		h_A[i] = i+1;
	}
	for(int i = 0; i < (colB*rowB); i++){
		h_B[i] = i+1;
	}

	//Copy Host memory to Device memory
	hipMemcpy(d_A,h_A, (colA*rowA)*sizeof(int), hipMemcpyHostToDevice);
  	hipMemcpy(d_B,h_B, (colB*rowB)*sizeof(int), hipMemcpyHostToDevice);

	//Declaring our 2D grid with 2D blocks 
	unsigned int gridRows = (rowA + BLOCK_SIZE - 1) / BLOCK_SIZE;
	unsigned int gridCols = (colB + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(gridCols, gridRows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	//Launch our Kernel
	multiMatrix<<<dimGrid, dimBlock>>>(d_A,d_B,d_C,colA,colB, rowA);

	//copy device results to host 
	hipMemcpy(h_C,d_C, (colB*rowA)*sizeof(int), hipMemcpyDeviceToHost);

	//print the results
	for(int i = 0; i < (rowA*colB); i++){
	  printf("%d ", h_C[i]);
	  if(((i+1) % colB) == 0)
		printf("\n");
	}

	// free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}

